#include "hip/hip_runtime.h"
#include "kernels.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "components/Config.h"

namespace eic {


namespace {

    __device__
    void renderCircle (float *s_canvas, const unsigned int width, const unsigned int height, float *g_shape_desc)
    {
        // Circle has the following representation
        // [0] = ShapeType::CIRCLE
        // [1] = R
        // [2] = G
        // [3] = B
        // [4] = alpha
        // [5] = center.x
        // [6] = center.y
        // [7] = radius

        // cv::Mat is organized in the h x w x 3 (01c) manner - we want to have the same
        int radius = g_shape_desc[7];
        int diameter = 2 * radius;
        int tl_x   = g_shape_desc[5] - radius;
        int tl_y   = g_shape_desc[6] - radius;

        for (int i = threadIdx.x; i < diameter*diameter; i += blockDim.x)
        {
            int y = int(i / diameter);
            int x = (i - (y * diameter));
            x += tl_x; y += tl_y;

            // Check the image bounds
            if (x >= 0 && y >= 0 && x < width && y < height)
            {
                s_canvas[3*width*y + 3*x + 0] = g_shape_desc[1]; // R
                s_canvas[3*width*y + 3*x + 1] = g_shape_desc[2]; // G
                s_canvas[3*width*y + 3*x + 2] = g_shape_desc[3]; // B
            }
        }
    }

}


// //////////////////////////////////////////////////////////////////////////////////////////////////////// //
// --------------------------------------------  CUDA KERNELS  -------------------------------------------- //
// //////////////////////////////////////////////////////////////////////////////////////////////////////// //

__global__
void populationFitness (__uint8_t *g_target, unsigned int width, unsigned int height, float *g_population,
                        unsigned int population_size, unsigned int chromozome_length, float *g_out_fitness,
                        float * g_canvas)
{
//    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    extern __shared__ float s_canvas[];  // width x height x 3 channels
    // Clear the whole canvas - set to 0
    for (int i = threadIdx.x; i < width*height; i += blockDim.x)
    {
        int row = i / width;
        int col = i - row*width;
        s_canvas[3*row*width + 3*col + 0] = 0;
        s_canvas[3*row*width + 3*col + 1] = 0;
        s_canvas[3*row*width + 3*col + 2] = 0;
    }

    // Chromozome id that is being rendered is given by the block id
    unsigned int ch_id = blockIdx.x;

    // Plot each shape in the chromozome
    float *g_chromozome = g_population + ch_id*(chromozome_length*DESC_LEN+1);
    float *g_shape_desc = g_chromozome + 1;
    for (int i = 0; i < chromozome_length; ++i)
    {
        // Render each shape
        if (ShapeType(g_shape_desc[0]) == ShapeType::CIRCLE)
        {
            renderCircle(s_canvas, width, height, g_shape_desc);
        }

        __syncthreads();

        g_shape_desc += DESC_LEN;
    }

    // Compute fitness


    for (int i = threadIdx.x; i < width*height; i += blockDim.x)
    {
        int row = i / width;
        int col = i - row*width;
        g_canvas[3*row*width + 3*col + 0] = s_canvas[3*row*width + 3*col + 0];
        g_canvas[3*row*width + 3*col + 1] = s_canvas[3*row*width + 3*col + 1];
        g_canvas[3*row*width + 3*col + 2] = s_canvas[3*row*width + 3*col + 2];
    }
}


}
