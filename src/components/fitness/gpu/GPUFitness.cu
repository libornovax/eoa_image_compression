#include "GPUFitness.h"

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include "components/Chromozome.h"
#include "check_error.h"
#include "kernels.h"


namespace eic {


void computeFitnessGPU (const std::vector<std::shared_ptr<Chromozome>> &chromozomes, bool write_channels)
{
    uchar* g_target; hipMalloc((void**)&g_target, 10*sizeof(uchar));
//    hipMemcpy(g_seq_in_out, seq.data(), seq.size()*sizeof(float), hipMemcpyHostToDevice);

    float* g_out_fitness; hipMalloc((void**)&g_out_fitness, 10*sizeof(float));

    float population[] = { 1.0f/*roi*/, 1.0f, 255.0f, 120.0f, 160.0f, 30.0f, 25.0f, 35.0f, 10.0f };
    float* g_population; hipMalloc((void**)&g_population, 11*sizeof(float));
    hipMemcpy(g_population, population, 11*sizeof(float), hipMemcpyHostToDevice);

    float* g_canvas; hipMalloc((void**)&g_canvas, 80*50*3*sizeof(float));


    populationFitness<<< 1, 64, 80*50*3*sizeof(float) >>>(g_target, 80, 50, g_population, 1, 1, g_out_fitness, g_canvas);

    cv::Mat canvas(50, 80, CV_32FC3);
    hipMemcpy(canvas.ptr<float>(), g_canvas, 80*50*3*sizeof(float), hipMemcpyDeviceToHost);

    std::cout << canvas << std::endl;

    canvas.convertTo(canvas, CV_8UC3);
    cv::cvtColor(canvas, canvas, CV_RGB2BGR);
    cv::imwrite("render.png", canvas);

    hipFree(g_target);
    hipFree(g_out_fitness);
    hipFree(g_population);
    hipFree(g_canvas);

    exit(EXIT_SUCCESS);
}


void computeFitnessGPU (const std::shared_ptr<Chromozome> &ch, bool write_channels)
{
    std::vector<std::shared_ptr<Chromozome>> chromozomes;
    chromozomes.push_back(ch);

    computeFitnessGPU(chromozomes, write_channels);
}


}

