#include "check_error.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


void handle_error(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        std::cout << hipGetErrorString(error) << " in " << file << " at line " << line << std::endl;
        exit(EXIT_FAILURE);
    }
}

